#include "hip/hip_runtime.h"
code = """

#include<iostream>
#include <cstdio>
#include <cstdlib>
#include<hip/hip_runtime.h>

using namespace std;

__global__ void minimum(int *input)
{
  int step_size = 1;
  int tid = threadIdx.x;
  int num_threads = blockDim.x;
  
  while(num_threads > 0)
  {
    if(tid < num_threads)
    {
      int first = tid*step_size*2;
      int second = first+step_size;
      if(input[second] < input[first])
        input[first] = input[second];
    }
    step_size *= 2;
    num_threads /= 2; 
  }

}


__global__ void maximum(int *input)
{
  int step_size = 1;
  int tid = threadIdx.x;
  int num_threads = blockDim.x;
  
  while(num_threads > 0)
  {
    if(tid < num_threads)
    {
      int first = tid*step_size*2;
      int second = first+step_size;
      if(input[second] > input[first])
        input[first] = input[second];
    }
    
    step_size *= 2;
    num_threads /= 2; 
  }

}

int main()
{
  int n;
  cin>>n;
  srand(n);
  int a[n];
  
  int min = 20000;
  
  for(int i=0;i<n;i++)
  {
    a[i] = rand()%20000;
    if(a[i] < min)
      min = a[i];
    cout<<a[i]<<"   ";
  }
  
  int size = n*sizeof(int);
  int *arr,result;
  
  hipMalloc(&arr,size);
  hipMemcpy(arr,a,size,hipMemcpyHostToDevice);
  
  minimum<<<1,n/2>>>(arr);
  
  hipMemcpy(&result,arr,sizeof(int),hipMemcpyDeviceToHost);
  cout<<"The minimum is :- "<<result;
  
  
  int *arr1,result1;
  
  hipMalloc(&arr1,size);
  hipMemcpy(arr1,a,size,hipMemcpyHostToDevice);
  
  maximum<<<1,n/2>>>(arr1);
  
  hipMemcpy(&result1,arr1,sizeof(int),hipMemcpyDeviceToHost);
  cout<<"The maximum is :- "<<result1;
    
  hipFree(arr);
  hipFree(arr1);
}

"""
