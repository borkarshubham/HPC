#include "hip/hip_runtime.h"
 int *arr2;
  int result2;
  hipMalloc(&arr2,size);
  hipMemcpy(arr2,a,size,hipMemcpyHostToDevice);
  
  std_dev<<<1,n/2>>>(arr2,avg);
  
  hipMemcpy(&result2,arr2,sizeof(int),hipMemcpyDeviceToHost);
  
  cout<<"****"<<result2;
  float result3 = result2/n;
  result3 = sqrt(result3);
  
  cout<<endl;
  cout<<"The standard deviation is :- "<<result3;
    
  hipFree(arr1);
  hipFree(arr2);



__global__ void std_dev(int *input,int a)
{
  int step_size = 1;
  int tid = threadIdx.x;
  int num_threads = blockDim.x;
  
  while(num_threads > 0)
  {
    if(tid < num_threads)
    {
      int first = tid*step_size*2;
      int second = first+step_size;
      
      input[first] = ((input[first]-a) * (input[first]-a));
      input[second] = ((input[second]- a) * (input[second] - a));
      input[first] += input[second];
    }
    
    step_size *= 2;
    num_threads /= 2; 
  }

}
