#include "hip/hip_runtime.h"
#include<iostream>
#include<hip/hip_runtime.h>

using namespace std;

__global__ void mat_mul(int *a,int *b,int *c,int m,int n,int p)
{
  int row = blockIdx.y*blockDim.y+threadIdx.y;
  int col = blockIdx.x*blockDim.x+threadIdx.x;
  int sum = 0;
  
  if(col < p && row < m)
  {
        for(int i=0;i<n;i++)
          sum += a[row*n+i]*b[i*p+col];
        
        c[row*p+col] = sum;    
  }
}

int main()
{
  int m = 5;
  int n = 4;
  int p = 3;
  
  int a[m*n],b[n*p],c[m*p];
    
   for(int i =0;i<m;i++)
   {
      for(int j=0;j<n;j++)
        a[i*n+j] = rand()%100;
   }
   
   for(int i =0;i<n;i++)
   {
      for(int j=0;j<p;j++)
        b[i*p+j] = rand()%100;
   }
  
  cout<<"The 1st matrix is :- "<<endl;
  for(int i =0;i<m;i++)
   {
      for(int j=0;j<n;j++)
        cout<<a[i*n+j]<<"  ";
       cout<<endl;
   }
   
   cout<<"The 2nd matrix is :- "<<endl;
   for(int i =0;i<n;i++)
   {
      for(int j=0;j<p;j++)
        cout<<b[i*p+j]<<"  ";
       cout<<endl;
   }
    
   int mat_size1 = m*n*sizeof(int);
   int mat_size2 = n*p*sizeof(int);
   int a_size = m*p*sizeof(int);
     
   int *d_v,*d_m,*d_a;
   
   hipMalloc(&d_v,mat_size1);
   hipMalloc(&d_m,mat_size2);
   hipMalloc(&d_a,a_size);
   
   hipMemcpy(d_v,a,mat_size1,hipMemcpyHostToDevice);
   hipMemcpy(d_m,b,mat_size2,hipMemcpyHostToDevice);
   
   dim3 dimGrid(1,1);
   dim3 dimBlock(16, 16);
   
   mat_mul<<<dimGrid,dimBlock>>>(d_v,d_m,d_a,m,n,p);
   
   hipMemcpy(c,d_a,a_size,hipMemcpyDeviceToHost);
   
   cout<<endl;
   cout<<"The result is :- "<<endl;   
   for(int i =0;i<m;i++)
   {
      for(int j=0;j<p;j++)
        cout<<c[i*p+j]<<"  ";
       cout<<endl;
   }
    
   hipFree(d_v);
   hipFree(d_m);
   hipFree(d_a);
   
}

"""
