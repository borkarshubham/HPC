#include "hip/hip_runtime.h"
code= """

#include<iostream>
#include<hip/hip_runtime.h>

using namespace std;

__global__ void vec_mat(int *a,int *b,int *c,int m,int n)
{
  int tid = blockIdx.x*blockDim.x+threadIdx.x;
  int sum = 0;
  if(tid < n)
  {
    if(tid < n)
    {
        for(int i=0;i<m;i++)
          sum += a[i]*b[i*n+tid];
         
         c[tid] = sum;
    
    }
    
  }
}

int main()
{
  int m = 3;
  int n = 4;
  
  int a[n],b[n*m],c[n];
  
  for(int i = 0;i<m;i++)
    a[i] = rand()%100;
    
   for(int i =0;i<m;i++)
   {
      for(int j=0;j<n;j++)
        b[i*n+j] = rand()%100;
   }
  
  cout<<"The vector is :- ";
  for(int i = 0;i<m;i++)
    cout<<a[i]<<"  ";
    
    
     cout<<endl;
    cout<<"The matrix is :- ";
    for(int i =0;i<m;i++)
   {
      for(int j=0;j<n;j++)
        cout<<b[i*n+j]<<"  ";
       cout<<endl;
   }
  
   int v_size = m*sizeof(int);
   int mat_size = n*m*sizeof(int);
   int a_size = n*sizeof(int);
     
   int *d_v,*d_m,*d_a;
   
   hipMalloc(&d_v,v_size);
   hipMalloc(&d_m,mat_size);
   hipMalloc(&d_a,a_size);
   
   hipMemcpy(d_v,a,v_size,hipMemcpyHostToDevice);
   hipMemcpy(d_m,b,mat_size,hipMemcpyHostToDevice);
   
   int n_threads = 256;
   
   vec_mat<<<n/256+1,n_threads>>>(d_v,d_m,d_a,m,n);
   
   hipMemcpy(c,d_a,a_size,hipMemcpyDeviceToHost);
   
   cout<<endl;
   cout<<"The result is :- ";   
   for(int j=0;j<n;j++)
      cout<<c[j]<<"  ";
    
   hipFree(d_v);
   hipFree(d_m);
   hipFree(d_a);
   
}

"""
