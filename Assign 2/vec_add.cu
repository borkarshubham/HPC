#include<iostream>
#include<hip/hip_runtime.h>

using namespace std;

__global__ void vec_add(int *a,int *b,int *c,int n)
{
  int tid = blockIdx.x*blockDim.x+threadIdx.x;
  
  if(tid < n)
  {
    c[tid] = a[tid] +b[tid];
  }
}

int main()
{
  int n = 100;
  
  int a[n],b[n],c[n];
  
  for(int i = 0;i<n;i++)
  {
    a[i] = rand()%100;
    b[i] = rand()%100;
  }
  
  cout<<"The 1st array is :- ";
  for(int i = 0;i<n;i++)
    cout<<a[i]<<"  ";
  
  cout<<endl;
  cout<<"The 2nd array is :- ";
  for(int i = 0;i<n;i++)
    cout<<b[i]<<"  ";
     
   int size = n*sizeof(int);  
     
   int *d_a,*d_b,*d_c;
   
   
   hipMalloc(&d_a,size);
   hipMalloc(&d_b,size);
   hipMalloc(&d_c,size);
   
   hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
   hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);
   
   int n_threads = 256;
   int grid_sz = (int) ceil((float)n/n_threads);
   
   vec_add<<<grid_sz,n_threads>>>(d_a,d_b,d_c,n);
   
   hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);
   
    cout<<endl;
    cout<<"The result is :- ";
    for(int i = 0;i<n;i++)
      cout<<c[i]<<"  ";
    
   hipFree(d_a);
   hipFree(d_b);
   hipFree(d_c);
   
 
}
